#include "hip/hip_runtime.h"
#include "md5.h"
#include <stdio.h>
#include <string.h>
#include <stdbool.h>
#include <time.h>
#include <pthread.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <atomic>

#define NUM_TARGET_HASHES 13
#define MAX_LEN 5
#define BLOCK_SIZE 256

// Target hashes, in the same format as before
const char *targetHashes[NUM_TARGET_HASHES] = {
"8d2855c1ed9047f655ce86f58ee3c4ed", "e3b0cf89846e85460d5f11caa2384acb",
"a7fb77d397a34877ab875e71f4bf236d", "683c11c14c7cb33c58c6220b5c1c7f51",
"3406e8d8f5244aeda183a1a71fbc2be6", "7013744587af48c686824a8b5015b075",
"9cf8e0a3a32e41039410f88f4ff101af", "c21483dc4ddc40a2a9c2a873af0b8f8c",
"175e915e8a5b4bcca345ee75b568ee63", "5c098bca9cb8464687d78c580b6a4a78",
"748ce7269dfd461787f3513d6d6e5352", "ae6fdb1ebdf7445eb0cd0ff9d111d34c",
"4ae57592275241e4912bc106ad470b04"
};

std::atomic<size_t> hashCount(0);  // Atomic counter for thread safety

// Check if the generated hash matches any target hash
__host__ bool isTargetHash(char *generatedHash) {
    for (int i = 0; i < NUM_TARGET_HASHES; i++) {
        if (strcmp(generatedHash, targetHashes[i]) == 0) {
            return true;
        }
    }
    return false;
}

// CUDA Error Check Macro
#define cudaCheckError() { \
    hipError_t e = hipGetLastError(); \
    if (e != hipSuccess) { \
        printf("CUDA Error: %s\n", hipGetErrorString(e)); \
        exit(EXIT_FAILURE); \
    } \
}

// Host MD5 brute-force function using CUDA
void bruteForceMD5_CUDA(char *charset, int maxLen) {
    int charsetSize = strlen(charset);
    
   // Allocate GPU memory and copy data
    char *d_charset;
    hipMalloc((void **)&d_charset, charsetSize * sizeof(char));
    cudaCheckError();
    hipMemcpy(d_charset, charset, charsetSize * sizeof(char), hipMemcpyHostToDevice);
    cudaCheckError();

    // Allocate other memory
    word **d_M;
    word *d_output;
    word T[65];  // MD5 T-table
    hipDeviceSetLimit(hipLimitMallocHeapSize, 1024 * 1024 * 1024);


    hipMalloc((void **)&d_M, sizeof(word *) * BLOCK_SIZE);
    cudaCheckError();
    hipMalloc((void **)&d_M, sizeof(word *) * BLOCK_SIZE);
    cudaCheckError();  // Ensure allocation is successful
    word *d_T;
    hipMalloc((void **)&d_T, 65 * sizeof(word));
    cudaCheckError();
    hipMemcpy(d_T, T, 65 * sizeof(word), hipMemcpyHostToDevice);
    cudaCheckError();
    
    // Compute the total number of combinations
    int totalCombinations = pow(charsetSize, maxLen);
    int numBlocks = (totalCombinations + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Launch kernel
    md5_bruteforce_Kernel<<<numBlocks, BLOCK_SIZE>>>(d_M, numBlocks, d_T, d_output);
    hipDeviceSynchronize();  // Ensure kernel execution is complete
    cudaCheckError();  // Check for errors after kernel execution

    // Copy output back to host
    word *output = (word *)malloc(sizeof(word) * 4 * BLOCK_SIZE);
    hipMemcpy(output, d_output, sizeof(word) * 4 * BLOCK_SIZE, hipMemcpyDeviceToHost);
    cudaCheckError();

    // Process results
    for (int i = 0; i < numBlocks * BLOCK_SIZE; i++) {
        char generatedHash[33];
        snprintf(generatedHash, sizeof(generatedHash), "%08x%08x%08x%08x", output[i * 4], output[i * 4 + 1], output[i * 4 + 2], output[i * 4 + 3]);
        
        if (isTargetHash(generatedHash)) {
            printf("Found match: %s\n", generatedHash);
        }
        hashCount.fetch_add(1);
    }

    // Free memory
    hipFree(d_charset);
    hipFree(d_M);
    hipFree(d_output);
    hipFree(d_T);
    free(output);
}

// Stats printing function running in a separate thread
void *printStats(void *arg) {
    size_t lastHashCount = 0;
    time_t lastTime = time(NULL); // Start the timer

    while (1) {
        sleep(5);  // Sleep for 5 seconds
        time_t currentTime = time(NULL);
        double elapsedTime = difftime(currentTime, lastTime);

        size_t currentHashCount = hashCount.load();
        size_t hashesProcessed = currentHashCount - lastHashCount;
        double hashesPerSecond = hashesProcessed / elapsedTime;
        printf("Hashes per second: %.2f\n", hashesPerSecond);

        // Update last values
        lastHashCount = currentHashCount;
        lastTime = currentTime;
    }

    return NULL; // Required for pthread functions
}

int main() {
    // Charset and maximum length for brute-forcing
    char charset[] = "abcdefghijklmnopqrstuvwxyz0123456789";
    int maxLen = MAX_LEN;

    // Start the stats printing in a separate thread
    pthread_t statsThread;
    pthread_create(&statsThread, NULL, printStats, NULL);

    // Start brute-forcing using CUDA
    bruteForceMD5_CUDA(charset, maxLen);

    // Wait for the stats thread to finish (it won't, but this is a clean way to join threads)
    pthread_join(statsThread, NULL);

    return 0;
}
